#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__global__
void distanceKernel(uchar4 *d_out, int w, int h, float2 pos1, float2 pos2, float2 pos3, float size3) {
  // pixel/thread mapping
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;

  if ((c >= w) || (r >= h)) return; // Check if within image bounds

  const int i = c + r*w; // 1D indexing
  const int dist1 = sqrtf((c - pos1.x)*(c - pos1.x) + 
	  (r - pos1.y)*(r - pos1.y));
  const int dist2 = sqrtf((c - pos2.x)*(c - pos2.x) +
	  (r - pos2.y)*(r - pos2.y));
  const int dist3 = sqrtf((c - pos3.x)*(c - pos3.x) +
	  (r - pos3.y)*(r - pos3.y)) * size3;

  d_out[i].x = clip(255 - dist1);
  d_out[i].y = clip(255 - dist2);
  d_out[i].z = clip(255 - dist3);
  d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, float2 pos1, float2 pos2, float2 pos3, float size3) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  distanceKernel<<<gridSize, blockSize>>>(d_out, w, h, pos1, pos2, pos3, size3);
}